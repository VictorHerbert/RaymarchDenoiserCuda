#include "hip/hip_runtime.h"
#include "window.cuh"

#include "imgui.h"
#include "backends/imgui_impl_glfw.h"
#include "backends/imgui_impl_opengl3.h"
#include <GLFW/glfw3.h>

#include <iostream>
#include <chrono>

#include "third_party/hip/hip_vector_types.h"
#include "image.cuh"
#include "filter.cuh"
#include "math.h"
#include "raymarch.cuh"

GLuint image_texture;
int display_w, display_h;

// Boilerplate from reference code
int window(){
    if (!glfwInit()){
        throw std::runtime_error("Failed to init GLFW");
        return -1;
    }
    GLFWwindow* window = glfwCreateWindow(1280, 720, "Denoiser", NULL, NULL);

    if (!window){
        throw std::runtime_error("Failed to create window");
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);
    glfwSwapInterval(1); // Enable vsync

    // Setup Dear ImGui
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;

    ImGui::StyleColorsDark();
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 130");

    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();

        glfwGetFramebufferSize(window, &display_w, &display_h);

        renderUI();

        glViewport(0, 0, display_w, display_h);
        glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        glfwSwapBuffers(window);
    }

    // Cleanup
    glDeleteTextures(1, &image_texture);
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();
    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}

bool raymarchCPU = false;
bool denoiseCPU = false;

int kersize = 2;
int depth = 1;
bool cpu = false;
float sigmaSpace = 0.5f;
float sigmaColor = 0.5f;
float sigmaAlbedo = 0.5f;
float sigmaNormal = 0.5;

float minSigma = 0.1;
float maxSigma = 1.0f;

int2 shape = {512, 512};

std::vector<float3> render(totalSize(shape));
std::vector<float3> normal(totalSize(shape));

auto previousFrameCheckpoint = std::chrono::high_resolution_clock::now();

Camera camera = {
    {0,-.5,-2}, // .pos 
    {0,0,1},    // forward
    {1,1},      // plane
    1           // dist
};



void renderUI() {
    using clock = std::chrono::high_resolution_clock;

    ImGui::NewFrame();
    ImGui::SetNextWindowPos(ImVec2(0, 0));

    const float aspect_ratio = (float)shape.x / (float)shape.y; // image ratio
    ImGui::SetNextWindowSizeConstraints(
        ImVec2(200, 200), 
        ImVec2(display_w, display_h),
        [](ImGuiSizeCallbackData* data) {
            float aspect = *((float*)data->UserData);
            float width = data->DesiredSize.x;
            float height = data->DesiredSize.y;

            if (width / height > aspect)
                width = height * aspect;
            else
                height = width / aspect;

            data->DesiredSize = ImVec2(width, height);
        },
        (void*)&aspect_ratio
    );

    ImGui::SetNextWindowSize(ImVec2(display_w, display_h));
    ImGui::PushStyleVar(ImGuiStyleVar_WindowRounding, 0.0f);
    ImGui::PushStyleVar(ImGuiStyleVar_WindowBorderSize, 0.0f);
    ImGui::Begin("Background", nullptr,
        ImGuiWindowFlags_NoDecoration |
        ImGuiWindowFlags_NoInputs |         // ignore mouse clicks
        ImGuiWindowFlags_NoBringToFrontOnFocus);

// -------------------------------------------------------------------------------

    auto frameCheckpoint = clock::now();
    std::chrono::duration<double, std::milli> frameTime = (frameCheckpoint - previousFrameCheckpoint);
    previousFrameCheckpoint = frameCheckpoint;

// -------------------------------------------------------------------------------

    static std::vector<Solid> solids = {
        {Light,     {0,1,0}, {.3,.3,.3}, {0,0,1}},

        {Box,      {0,-1,0},  {1,0.01,1},   {1,1,1}},
        {Box,    {0,0.5,0},   {1,0.01,1},   {1,1,1}},
        {Box,   {0,0,0.5},   {1,1,0.01},   {1,1,1}},
        {Box,   {-0.5,0,0},  {0.01,1,1},   {1,0,0}},
        {Box,  {0.5,0,0},   {0.01,1,1},   {0,1,0}},

        {Box,       {-0.15,-0.25,0.15},  {0.2,0.25,0.2},   {0,0,1}},
        {Box,       {0.2,-0.35,-0.1},    {0.15,0.35,0.15}, {1,1,0}}
    };
    
    static CudaVector<Solid> scene(solids.size());
    static CudaVector<float3> render(totalSize(shape)), normal(totalSize(shape)), albedo(totalSize(shape));
 
    static std::vector<float3> render_cpu(totalSize(shape));

    static GLuint img_texture;

    static bool update = true;

    if(raymarchCPU){

    }
    else {
        scene.copy(solids);
        raymarchSceneGPU(camera, {scene.size, scene.data}, {shape, render.data, normal.data, albedo.data, nullptr});
        //hipMemcpy(render_cpu.data(), render.data, sizeof(float3)*totalSize(shape), hipMemcpyDeviceToHost);
        img_texture = textureFromBuffer(render_cpu.data(), shape);
    }
    


// -------------------------------------------------------------------------------
    std::chrono::duration<double, std::milli> processingTime = clock::now() - frameCheckpoint;
// -------------------------------------------------------------------------------

    ImVec2 avail = ImGui::GetContentRegionAvail();
    ImGui::Image((ImTextureID)(intptr_t) img_texture, avail, ImVec2(0, 0), ImVec2(1, 1)); // fit image
    ImGui::End();
    ImGui::PopStyleVar(2);

    // --- Menu window remains unchanged ---
    ImGui::SetNextWindowPos(ImVec2(800, 30), ImGuiCond_Once);
    ImGui::SetNextWindowSize(ImVec2(300, 0));

// -------------------------------------------------------------------------------

    ImGui::Begin("Menu");

    ImGui::SeparatorText("Performance");
    ImGui::Text("FPS: %.1f", 1000/frameTime.count());
    ImGui::Text("UI Overhead: %.2f %%", (1-processingTime.count()/frameTime.count())*100);
    ImGui::Text("Frame time: %.2f/%.2f ms", processingTime.count(), frameTime.count());

    ImGui::SeparatorText("Viewport");
    ImGui::InputInt2("Resolution", (int*) &shape, .1);

    const char* items[] = { "Render", "Denoised", "Albedo", "Normal"};
    static int item_selected_idx = 0;

    const char* combo_preview_value = items[item_selected_idx];
    if (ImGui::BeginCombo("Channel", combo_preview_value)){
        for (int n = 0; n < IM_ARRAYSIZE(items); n++){
            if (ImGui::Selectable(items[n], item_selected_idx == n))
                item_selected_idx = n;
        }
        ImGui::EndCombo();
    }

    ImGui::SeparatorText("Scene");

    if(ImGui::CollapsingHeader("Camera")){
        ImGui::DragFloat3("Pos", (float*) &camera.pos, .1);
        ImGui::DragFloat3("Dir", (float*) &camera.forward, .1);
        ImGui::DragFloat2("Plane", (float*) &camera.plane, .1);
        ImGui::DragFloat("Dist", (float*) &camera.dist, .1);
    }
    camera.forward = normalize(camera.forward);
    float dummy []= {.5,.5,.5};

    const char* objTypeNames[] = {"Light", "Sphere", "Box"};

    for(int i = 0; i < scene.size; i++){
        ImGui::PushID(i);
        if(ImGui::CollapsingHeader(objTypeNames[solids[i].type])){
            ImGui::DragFloat3("Pos", (float*) &solids[i].pos, .1);
            ImGui::DragFloat3("Scale", (float*) &solids[i].scale, .1);
            ImGui::ColorEdit3("Color", (float*) &solids[i].col, ImGuiColorEditFlags_Float);
        }
        ImGui::PopID();
    }

    ImGui::SeparatorText("Setup");

    if(ImGui::CollapsingHeader("Raymarch")){
        ImGui::Checkbox("CPU Raymarch", &raymarchCPU);
        ImGui::Checkbox("Anti-aliasing", (bool*) dummy);
    }
    if(ImGui::CollapsingHeader("Denoising")){
        ImGui::Checkbox("CPU Denoise", &denoiseCPU);
        ImGui::Spacing();
        ImGui::SliderInt("Iterations", &depth, 0, 10);
        ImGui::Spacing();
        ImGui::SliderFloat("Sigma Color", &sigmaColor, minSigma, maxSigma);
        ImGui::SliderFloat("Sigma Albedo", &sigmaAlbedo, minSigma, maxSigma);
        ImGui::SliderFloat("Sigma Normal", &sigmaNormal, minSigma, maxSigma);
    }

    ImGui::End();
    ImGui::Render();
}

GLuint textureFromBuffer(float3* image, int2 shape){
    GLuint tex_id;
    glGenTextures(1, &tex_id);
    glBindTexture(GL_TEXTURE_2D, tex_id);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB32F, shape.x, shape.y, 0, GL_RGB, GL_FLOAT, image);

    return tex_id;
}