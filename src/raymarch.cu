#include "hip/hip_runtime.h"
#include "raymarch.cuh"

#include "matrix.cuh"
#include "third_party/hip/hip_vector_types.h"


const int MAX_STEPS = 100;
const float MAX_DIST = 1e4;
const float SURF_DIST = 1e-6;

float2 operator/(const int2& a, const int2& b) {
    return make_float2((float) a.x/b.x, (float) a.y/b.y);
}

float3 viewportToWorld(int2 pos, int2 shape, Camera camera) {
    float3 worldUp = {0,1,0};
    float3 right = normalize(cross(camera.forward, worldUp));
    float3 up    = normalize(cross(right, camera.forward));

    float u = (2.0f * pos.x / (float)shape.x - 1.0f);
    float v = (2.0f * pos.y / (float)shape.y - 1.0f);

    float3 planeCenter = camera.forward * camera.dist;
    float3 posInPlane = (u * camera.plane.x * 0.5f) * right + (v * camera.plane.y * 0.5f) * up;

    return camera.pos + planeCenter + posInPlane;
}


float sdfSphere(float3 pos, float r){
    return length(pos)-r;
}

float sdfPlane(float3 pos){
    return -pos.y;
}


RenderData raymarchPoint(float3 pos, Scene scene){
    RenderData data;
    data.depth = MAX_DIST;
    
    for(int i = 0; i < scene.size; i++){
        Solid solid = scene.solids[i];
        float currDist;

        switch (solid.type){
        case Sphere:
            currDist = sdfSphere(pos, solid.scale.x);
            break;
        case Box:
           //currDist = sdfBox(pos);
        default:
            break;
        }

        if(currDist < data.depth){
            data.depth = currDist;
            data.id = i;
            if(i == 1)
                data.col = make_float3(1,0,0);
            if(i == 0)
                data.col = make_float3(0,0,1);
        }
    }
    return data;
}

float3 raymarchNormal(float3 p, Scene scene) {
	float d = raymarchPoint(p, scene).depth;
    float e = .01;
    
    float3 n = d - make_float3(
        raymarchPoint(p-make_float3(e,0,0), scene).depth,
        raymarchPoint(p-make_float3(0,e,0), scene).depth,
        raymarchPoint(p-make_float3(0,0,e), scene).depth
    );
    
    return normalize(n);
}

RenderData raymarchRay(Ray ray, Scene scene){
    RenderData data;
    float distTotal = 0;
    float distStep = 0;
    float3 posCurr;

    for(int i = 0; i < MAX_STEPS; i++) {
        distTotal += distStep;
        if(distTotal > MAX_DIST){
            data.id = -1;
            data.col = {0,0,0};
            data.normal = {0,0,0};
            break;
        }

    	posCurr = ray.origin + ray.direction * distTotal;
        data = raymarchPoint(posCurr, scene);
        distStep = data.depth;
        
        if(distStep < SURF_DIST) break;
    }

    if(data.id != -1)
        data.normal = raymarchNormal(posCurr, scene);

    return data;
}

void raymarchSceneCPU(Camera camera, Scene scene, Framebuffer framebuffer){
    int2 pixelPos = {0,0};
    for(pixelPos.x = 0; pixelPos.x < framebuffer.shape.x; pixelPos.x++){
        for(pixelPos.y = 0; pixelPos.y < framebuffer.shape.y; pixelPos.y++){
            raymarchScenePixel(pixelPos, camera, scene, framebuffer);
        }
    }
}


void raymarchSceneGPU(Camera camera, Scene scene, Framebuffer framebuffer){
    dim3 blockSize(16, 16);
    dim3 gridSize((framebuffer.shape.x + 15) / 16, (framebuffer.shape.y + 15) / 16);

    raymarchSceneKernel<<<gridSize,blockSize>>>(camera, scene, framebuffer);
    hipDeviceSynchronize();
}


KERNEL void raymarchSceneKernel(Camera camera, Scene scene, Framebuffer framebuffer){
    int2 pos = {
        blockIdx.x * blockDim.x + threadIdx.x,
        blockIdx.y * blockDim.y + threadIdx.y
    };

    if(pos.x >= framebuffer.shape.x || pos.y >= framebuffer.shape.y)
        return;
    
    raymarchScenePixel(pos, camera, scene, framebuffer);
}

KFUNC void raymarchScenePixel(int2 pixelPos, Camera camera, Scene scene, Framebuffer framebuffer){
    float3 worldPos = viewportToWorld(pixelPos, framebuffer.shape, camera);
    Ray ray = {camera.pos, normalize(worldPos - camera.pos)};
    RenderData data = raymarchRay(ray, scene);
    framebuffer.render[index(pixelPos, framebuffer.shape)] = data.col;
    framebuffer.normal[index(pixelPos, framebuffer.shape)] = data.normal;

    framebuffer.render[index(pixelPos, framebuffer.shape)] = {255, 0, 0};
}
